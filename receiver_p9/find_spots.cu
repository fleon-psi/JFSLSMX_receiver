#include "hip/hip_runtime.h"
/*
 * Copyright 2020 Paul Scherrer Institute
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Acknowledgements K. Diederichs (U. Konstanz)

#include <sys/types.h>
#include <sys/socket.h>

#include <iostream>
#include "JFReceiver.h"

// modules are stacked two vertically
// 67 (modules 6 and 7)
// 45
// 32
// 01
// --> but this part of app cares about four top/bottom modules
// --> so one chunk will be 67 and another 45 (or resp. 32 and 01)
#define FRAGMENT_SIZE_16 ((NMODULES/2) * COLS * LINES * sizeof(int16_t))

// CUDA calculation streams
hipStream_t stream[NCUDA_STREAMS];

// GPU kernel to find strong pixels
template<typename T>
__global__ void find_spots_colspot(T *in, strong_pixel *out, float strong, int N) {
     if (blockIdx.x * blockDim.x + threadIdx.x < N) {
        // Threshold for signal^2 / var
        // To avoid division (see later) N/(N-1) factor is included already in the threshold
        float threshold = strong * strong * (float)((2*NBX+1) * (2*NBY+1)) / (float) ((2*NBX+1) * (2*NBY+1)-1);

        // One thread is 514 lines or 2 modules (in 2x2 configuration)
        // line0 points to the module/frame
        size_t line0 = (blockIdx.x * blockDim.x + threadIdx.x) * LINES;

        // Location of the first strong pixel in the output array 
        size_t strong_id0 = (blockIdx.x * blockDim.x + threadIdx.x) * MAX_STRONG;
        size_t strong_id = 0;

        // Sum and sum of squares of (2*NBY+1) vertical elements 
        // These are updated after each line is finished
        // 64-bit integer guarantees calculations are made without rounding errors
        int64_t sum_vert[COLS];
        int64_t sum2_vert[COLS];

        // Precalculate squares for first 2*NBY+1 lines
        for (int col = 0; col < COLS; col++) {
            int64_t tmp = in[(line0) * COLS + col];
            sum_vert[col]  = tmp;
            sum2_vert[col] = tmp*tmp;
        }
 
        for (size_t line = 1; line < 2*NBY+1; line++) {
            for (int col = 0; col < COLS; col++) {
                int64_t tmp = in[(line0 + line) * COLS + col];
                sum_vert[col]  += tmp;
                sum2_vert[col] += tmp*tmp;
            }
        }

        // do calculations for lines NBY to MODULE_LINES - NBY
        for (int16_t line = NBY; line < LINES - NBY; line++) {

            // sum and sum of squares for (2*NBX+1) x (2*NBY+1) elements
            int64_t sum  = sum_vert[0]; // Should be divided (float)((2*NBX+1) * (2*NBY+1));
            int64_t sum2 = sum2_vert[0];

            for (int i = 1; i < 2*NBX+1; i ++) {
                sum  += sum_vert[i];
                sum2 += sum2_vert[i];
            }

            for (int16_t col = NBX; col < COLS - NBX; col++) {
                // At all cost division and sqrt must be avoided
                // as performance penalty is significant (2x drop)
                // instead, constants ((2*NBX+1) * (2*NBY+1)) and ((2*NBX+1) * (2*NBY+1)-1)
                // are included in the threshold
                int64_t var = (2*NBX+1) * (2*NBY+1) * sum2 - (sum * sum); // This should be divided by ((2*NBX+1) * (2*NBY+1)-1)*((2*NBX+1) * (2*NBY+1))
                int64_t in_minus_mean = in[(line0 + line)*COLS+col] * ((2*NBX+1) * (2*NBY+1)) - sum; // Should be divided by ((2*NBX+1) * (2*NBY+1));

                if ((in_minus_mean > (2*NBX+1) * (2*NBY+1)) && // pixel value is larger than mean
                    (in[(line0 + line)*COLS+col] > 0) && // pixel is not bad pixel and is above 0
                    (in_minus_mean * in_minus_mean > var * threshold)) {
                       // Save line, column and photon count in output table
                       out[strong_id0+strong_id].line = line;
                       out[strong_id0+strong_id].col = col;
                       out[strong_id0+strong_id].photons = in_minus_mean;
                       strong_id = (strong_id + 1 ) % MAX_STRONG;
                    }

                // Updated value of sum and sum2
                // For last column - these need not to be calculated
                if (col < COLS - NBX - 1) {
                   sum += sum_vert[col + NBX + 1] - sum_vert[col - NBX];
                   sum2 += sum2_vert[col + NBX + 1] - sum2_vert[col - NBX];

                }
            }
            // Shift sum_vert and sum2_vert by one line
            if (line < LINES - NBY - 1) {
                for (int col = 0; col < COLS; col++) {
                    int64_t tmp_sum  = (int64_t)in[(line0+line+NBY+1) * COLS + col] + (int64_t)in[(line0 + line-NBY) * COLS + col];
                    int64_t tmp_diff = (int64_t)in[(line0+line+NBY+1) * COLS + col] - (int64_t)in[(line0 + line-NBY) * COLS + col];
                    sum_vert[col]  += tmp_diff;
                    sum2_vert[col] += tmp_sum * tmp_diff; // in[(line0+line+NBY+1) * MODULE_COLS + col]^2 - in[(line0 + line-NBY) * MODULE_COLS + col]^2
                }
            }
        }
        // Mark, where useful data and in output table
        out[strong_id0+strong_id].line = -1;
        out[strong_id0+strong_id].col = -1;
        out[strong_id0+strong_id].photons = strong_id;
   }
}

char *gpu_data;
strong_pixel *gpu_out;

int setup_gpu(int device) {
    // Set device
    hipSetDevice(device);

    // Register image buffer as HW pinned (this is also registered by IB verbs)
    hipError_t err = hipHostRegister(ib_buffer, ib_buffer_size, hipHostRegisterPortable);
    if (err != hipSuccess) {
         std::cerr << "GPU: Register error " << hipGetErrorString(err) << " addr " << ib_buffer << " size " << ib_buffer_size << std::endl;
         return 1;
    }

    // NIMAGES_PER_STREAM * FRAGMENT_SIZE is the same for 16 and 32-bit image
    // there is half images per stream, but twice in size
    // Initialize input memory on GPU
    size_t gpu_data_size = NCUDA_STREAMS * NIMAGES_PER_STREAM * FRAGMENT_SIZE_16;
    err = hipMalloc((void **) &gpu_data, gpu_data_size);
    if (err != hipSuccess) {
         std::cerr << "GPU: Mem alloc. error (data) " <<  gpu_data_size / 1024 / 1024 << std::endl;
         return 1;
    }

    // Initialize output memory as GPU/CPU unified memory
    err = hipMallocManaged((void **) &gpu_out, NCUDA_STREAMS * NIMAGES_PER_STREAM * 2 * MAX_STRONG * sizeof(strong_pixel)); // frame is divided into 2 vertical slices
    if (err != hipSuccess) {
         std::cerr << "GPU: Mem alloc. error (output)" << std::endl;
         return 1;
    }

    // Create computing streams
    for (int i = 0; i < NCUDA_STREAMS; i++) {
        err = hipStreamCreate(&stream[i]);
        if (err != hipSuccess) {
            std::cerr << "GPU: Stream create error" << std::endl;
            return 1;
        }
    }

    // Setup synchronization
    for (int i = 0; i < NCUDA_STREAMS*CUDA_TO_IB_BUFFER; i++) {
            pthread_mutex_init(cuda_stream_ready_mutex+i, NULL);
            pthread_cond_init(cuda_stream_ready_cond+i, NULL);
            pthread_mutex_init(writer_threads_done_mutex+i, NULL);
            pthread_cond_init(writer_threads_done_cond+i, NULL);
    }
    return 0;
}

int close_gpu() {
    hipFree(gpu_out);
    hipFree(gpu_data);
    hipError_t err = hipHostUnregister(ib_buffer);
    for (int i = 0; i < NCUDA_STREAMS; i++)
        err = hipStreamDestroy(stream[i]);

    // Close synchronization
    for (int i = 0; i < NCUDA_STREAMS*CUDA_TO_IB_BUFFER; i++) {
            pthread_mutex_destroy(cuda_stream_ready_mutex+i);
            pthread_cond_destroy(cuda_stream_ready_cond+i);
            pthread_mutex_destroy(writer_threads_done_mutex+i);
            pthread_cond_destroy(writer_threads_done_cond+i);
    }

    return 0;
}

void *run_gpu_thread(void *in_threadarg) {
    ThreadArg *arg = (ThreadArg *) in_threadarg;

    // GPU device is valid on per-thread basis, so every thread needs to set it
    hipSetDevice(receiver_settings.gpu_device);

    // NIMAGES_PER_STREAM is defined for 16-bit image, so it needs to be adjusted for 32-bit
    size_t images_per_stream = NIMAGES_PER_STREAM * 2 / experiment_settings.pixel_depth;
    size_t fragment_size = ((NMODULES/2) * COLS * LINES * experiment_settings.pixel_depth);

    size_t total_chunks = experiment_settings.nimages_to_write / images_per_stream;
    // Account for leftover
    if (experiment_settings.nimages_to_write - total_chunks * images_per_stream > 0)
           total_chunks++;

    size_t thread_id = arg->ThreadID;

    hipEvent_t event_mem_copied;
    hipEventCreate (&event_mem_copied);

    for (size_t chunk = thread_id;
         chunk < total_chunks;
         chunk += NCUDA_STREAMS) {

         std::vector<spot_t> spots;

         size_t ib_slice = chunk % (NCUDA_STREAMS*CUDA_TO_IB_BUFFER);

         size_t images = experiment_settings.nimages_to_write - chunk * images_per_stream;
         if (images > images_per_stream) images = images_per_stream;

         pthread_mutex_lock(writer_threads_done_mutex+ib_slice);
         // Wait till everyone is done
         while (writer_threads_done[ib_slice] < receiver_settings.compression_threads)
             pthread_cond_wait(writer_threads_done_cond+ib_slice, 
                               writer_threads_done_mutex+ib_slice);
         // Restore full values and continue
         writer_threads_done[ib_slice] = 0;
         pthread_mutex_unlock(writer_threads_done_mutex+ib_slice);

         // Here all writting is done, but it is guarranteed not be overwritten

         // Copy frames to GPU memory
         hipError_t err;
         err = hipMemcpyAsync(gpu_data + thread_id * images_per_stream * fragment_size, 
               ib_buffer + ib_slice * images_per_stream * fragment_size,
               images * fragment_size,
               hipMemcpyHostToDevice, stream[thread_id]);
         if (err != hipSuccess) {
             std::cerr << "GPU: memory copy error for slice " << thread_id << "/" << ib_slice << "frames: " << images << "(" << hipGetErrorString(err) << ")" << std::endl;
             pthread_exit(0);
         }

         hipEventRecord (event_mem_copied, stream[thread_id]);

         // Start GPU kernel
         if (experiment_settings.pixel_depth == 2)
             find_spots_colspot<int16_t> <<<images_per_stream * 2 / 32, 32, 0, stream[thread_id]>>>
                 ((int16_t *) (gpu_data + thread_id * images_per_stream * fragment_size),
                  gpu_out + thread_id * images_per_stream * 2 * MAX_STRONG,
                  experiment_settings.strong_pixel, images * 2);
         else
             find_spots_colspot<int32_t> <<<images_per_stream * 2 / 32, 32, 0, stream[thread_id]>>>
                 ((int32_t *) (gpu_data + thread_id * images_per_stream * fragment_size),
                  gpu_out + thread_id * images_per_stream * 2 * MAX_STRONG,
                  experiment_settings.strong_pixel, images * 2);

         // After data are copied, one can release buffer
         err = hipEventSynchronize(event_mem_copied);
         if (err != hipSuccess) {
             std::cerr << "GPU: memory copy error" << std::endl;
             pthread_exit(0);
         }

         // Broadcast to everyone waiting, that buffer can be overwritten by next iteration
         pthread_mutex_lock(cuda_stream_ready_mutex+ib_slice);
         cuda_stream_ready[ib_slice] = chunk + NCUDA_STREAMS*CUDA_TO_IB_BUFFER;
         pthread_cond_broadcast(cuda_stream_ready_cond+ib_slice);
         pthread_mutex_unlock(cuda_stream_ready_mutex+ib_slice);

         // Ensure kernel has finished
         err = hipStreamSynchronize(stream[thread_id]);
         if (err != hipSuccess) {
             std::cerr << "GPU: execution error" << std::endl;
             pthread_exit(0);
         }

         // Analyze results to find spots
         // gpu_out is in unified memory and doesn't need to be explicitly copied to CPU
         analyze_spots(gpu_out + thread_id * images_per_stream * 2 * MAX_STRONG, spots, experiment_settings.connect_spots_between_frames, images, chunk * images_per_stream);

         // Send spots found by spot finder via TCP/IP
         pthread_mutex_lock(&accepted_socket_mutex);
         size_t spot_data_size = spots.size();
         send(accepted_socket, &spot_data_size, sizeof(size_t), 0);
         send(accepted_socket, spots.data(), spot_data_size * sizeof(spot_t), 0);
         pthread_mutex_unlock(&accepted_socket_mutex);
    }
    hipEventDestroy (event_mem_copied);
    pthread_exit(0);
}

