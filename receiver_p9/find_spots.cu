#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <map>
#include <vector>
#include <pthread.h>
#include "JFReceiver.h"

// Maximum number of strong pixel in 2 veritcal modules
// if there are more pixels, these will be overwritten
// in ring buffer fashion
#define MAX_STRONG 16384L

// Size of bounding box for pixel
#define NBX 3
#define NBY 3

// TODO - this should be in common header
#define COLS (2*1030L)
#define LINES (514L)
#define NCARDS 2

// modules are stacked two vertically
// 67 (modules 6 and 7)
// 45
// 32
// 01
// --> but this part of app cares about four top/bottom modules
// --> so one chunk will be 67 and another 45 (or resp. 32 and 01)
#define FRAME_SIZE ((NMODULES/2) * COLS * LINES * sizeof(int16_t))

// CUDA calculation streams
hipStream_t stream[NCUDA_STREAMS];

// GPU kernel to find strong pixels
template<typename T>
__global__ void find_spots_colspot(T *in, strong_pixel *out, float strong, int N) {
     if (blockIdx.x * blockDim.x + threadIdx.x < N) {
        // Threshold for signal^2 / var
        // To avoid division (see later) N/(N-1) factor is included already in the threshold
        float threshold = strong * strong * (float)((2*NBX+1) * (2*NBY+1)) / (float) ((2*NBX+1) * (2*NBY+1)-1);

        // One thread is 514 lines or 2 modules (in 2x2 configuration)
        // line0 points to the module/frame
        size_t line0 = (blockIdx.x * blockDim.x + threadIdx.x) * LINES;

        // Location of the first strong pixel in the output array 
        size_t strong_id0 = (blockIdx.x * blockDim.x + threadIdx.x) * MAX_STRONG;
        size_t strong_id = 0;

        // Sum and sum of squares of (2*NBY+1) vertical elements 
        // These are updated after each line is finished
        // 64-bit integer guarantees calculations are made without rounding errors
        int64_t sum_vert[COLS];
        int64_t sum2_vert[COLS];

        // Precalculate squares for first 2*NBY+1 lines
        for (int col = 0; col < COLS; col++) {
            sum_vert[col]  = in[(line0) * COLS + col];
            sum2_vert[col] = in[(line0) * COLS + col]*in[(line0) * COLS + col];
        }
 
        for (size_t line = 1; line < 2*NBY+1; line++) {
            for (int col = 0; col < COLS; col++) {
                sum_vert[col]  += in[(line0 + line) * COLS + col];
                sum2_vert[col] += in[(line0 + line) * COLS + col] * in[(line0 + line) * COLS + col];
            }
        }

        // do calculations for lines NBY to MODULE_LINES - NBY
        for (size_t line = NBY; line < LINES - NBY; line++) {

            // sum and sum of squares for (2*NBX+1) x (2*NBY+1) elements
            int64_t sum  = sum_vert[0];
            int64_t sum2 = sum2_vert[0];

            for (int i = 1; i < 2*NBX+1; i ++) {
                sum  += sum_vert[i];
                sum2 += sum2_vert[i];
            }

            for (int col = NBX; col < COLS - NBX; col++) {

                // At all cost division and sqrt must be avoided
                // as performance penalty is significant (2x drop)
                // instead, constants ((2*NBX+1) * (2*NBY+1)) and ((2*NBX+1) * (2*NBY+1)-1)
                // are included in the threshold
                float var = (2*NBX+1) * (2*NBY+1) * sum2 - (sum * sum); // This should be divided by (float) ((2*NBX+1) * (2*NBY+1)-1)

                float mean = sum; // Should be divided (float)((2*NBX+1) * (2*NBY+1));
                float in_minus_mean = in[(line0 + line)*COLS+col] * (float)((2*NBX+1) * (2*NBY+1)) - mean; // Should be divided (float)((2*NBX+1) * (2*NBY+1));

                if ((in_minus_mean > 0.0f) && // pixel value is larger than mean
                    (mean > 0.0f) &&          // mean is larger than zero (no bad pixels)
                    (in[(line0 + line)*COLS+col] > 0) && // pixel is not bad pixel and is above 0
                    (in_minus_mean * in_minus_mean > var * threshold)) {
                       // Save line, column and photon count in output table
                       out[strong_id0+strong_id].line = line;
                       out[strong_id0+strong_id].col = col;
                       out[strong_id0+strong_id].photons = in[(line0 + line)*COLS+col];
                       strong_id = (strong_id + 1 ) % MAX_STRONG;
                    }

                // Updated value of sum and sum2
                // For last column - these need not to be calculated
                if (col < COLS - NBX - 1) {
                   sum += sum_vert[col + NBX + 1] - sum_vert[col - NBX];
                   sum2 += sum2_vert[col + NBX + 1] - sum2_vert[col - NBX];

                }
            }
            // Shift sum_vert and sum2_vert by one line
            if (line < LINES - NBY - 1) {
                for (int col = 0; col < COLS; col++) {
                    int64_t tmp_sum  = (int64_t)in[(line0+line+NBY+1) * COLS + col] + (int64_t)in[(line0 + line-NBY) * COLS + col];
                    int64_t tmp_diff = (int64_t)in[(line0+line+NBY+1) * COLS + col] - (int64_t)in[(line0 + line-NBY) * COLS + col];
                    sum_vert[col]  += tmp_diff;
                    sum2_vert[col] += tmp_sum * tmp_diff; // in[(line0+line+NBY+1) * MODULE_COLS + col]^2 - in[(line0 + line-NBY) * MODULE_COLS + col]^2
                }
            }
        }
        // Mark, where useful data and in output table
        out[strong_id0+strong_id].line = -1;
        out[strong_id0+strong_id].col = -1;
        out[strong_id0+strong_id].photons = strong_id;
   }
}

int16_t *gpu_data16;
int32_t *gpu_data32;
strong_pixel *gpu_out;

int setup_gpu(int device) {
    // Set device
    hipSetDevice(device);

    // Register image buffer as HW pinned (this is also registered by IB verbs)
    hipError_t err = hipHostRegister(ib_buffer, ib_buffer_size, hipHostRegisterPortable);
    if (err != hipSuccess) {
         std::cerr << "GPU: Register error " << hipGetErrorString(err) << " addr " << ib_buffer << " size " << ib_buffer_size << std::endl;
         return 1;
    }

    // Initialize input memory on GPU
    size_t gpu_data16_size = NCUDA_STREAMS * NIMAGES_PER_STREAM * FRAME_SIZE;
    err = hipMalloc((void **) &gpu_data16, gpu_data16_size);
    if (err != hipSuccess) {
         std::cerr << "GPU: Mem alloc. error (data) " <<  gpu_data16_size / 1024 / 1024 << std::endl;
         return 1;
    }

    // Initialize output memory on GPU
    err = hipMalloc((void **) &gpu_out, NCUDA_STREAMS * NIMAGES_PER_STREAM * 2 * MAX_STRONG * sizeof(strong_pixel)); // frame is divided into 2 vertical slices
    if (err != hipSuccess) {
         std::cerr << "GPU: Mem alloc. error (output)" << std::endl;
         return 1;
    }

    // Create computing streams
    for (int i = 0; i < NCUDA_STREAMS; i++) {
        err = hipStreamCreate(&stream[i]);
        if (err != hipSuccess) {
            std::cerr << "GPU: Stream create error" << std::endl;
            return 1;
        }
    }

    // Setup synchronization
    for (int i = 0; i < NCUDA_STREAMS*CUDA_TO_IB_BUFFER; i++) {
            pthread_mutex_init(cuda_stream_ready_mutex+i, NULL);
            pthread_cond_init(cuda_stream_ready_cond+i, NULL);
            pthread_mutex_init(writer_threads_done_mutex+i, NULL);
            pthread_cond_init(writer_threads_done_cond+i, NULL);
    }
    return 0;
}

int close_gpu() {
    hipFree(gpu_out);
    hipFree(gpu_data16);
    hipError_t err = hipHostUnregister(ib_buffer);
    for (int i = 0; i < NCUDA_STREAMS; i++)
        err = hipStreamDestroy(stream[i]);

    // Close synchronization
    for (int i = 0; i < NCUDA_STREAMS*CUDA_TO_IB_BUFFER; i++) {
            pthread_mutex_destroy(cuda_stream_ready_mutex+i);
            pthread_cond_destroy(cuda_stream_ready_cond+i);
            pthread_mutex_destroy(writer_threads_done_mutex+i);
            pthread_cond_destroy(writer_threads_done_cond+i);
    }

    return 0;
}

// CPU part
// (recursion will not fit well to GPU)
// Constructing spot from strong pixels

// Adds two spot measurements
void merge_spots(spot_t &spot1, const spot_t spot2) {
        spot1.x = spot1.x + spot2.x;
        spot1.y = spot1.y + spot2.y;
        spot1.z = spot1.z + spot2.z;
        spot1.photons = spot1.photons + spot2.photons;
        spot1.pixels = spot1.pixels + spot2.pixels;
}

// If spots come from two different frames, depth needs to be incremented
void merge_spots_new_frame(spot_t &spot1, const spot_t spot2) {
        spot1.x = spot1.x + spot2.x;
        spot1.y = spot1.y + spot2.y;
        spot1.z = spot1.z + spot2.z;
        spot1.photons = spot1.photons + spot2.photons;
        spot1.pixels = spot1.pixels + spot2.pixels;
        spot1.depth = spot1.depth + spot2.depth + 1;
}

typedef std::pair<int16_t, int16_t> coordxy_t; // This is simply (x, y)
typedef std::map<coordxy_t, uint64_t> strong_pixel_map_t;
// This is mapping (x,y) --> intensity
// it allows to find if there is spot in (x,y) in log time
typedef std::vector<strong_pixel_map_t> strong_pixel_maps_t;
// There is one map per 1/2 frame

// Creates a continous spot
// strong pixels are loaded into dictionary (one dictionary per frame)
// and routine checks if neighboring pixels are also in dictionary (likely in log(N) time)
spot_t add_pixel(strong_pixel_maps_t &strong_pixel_maps, size_t i, strong_pixel_map_t::iterator &it, bool connect_frames) {
    spot_t ret_value;

    uint64_t photons = it->second;
    int16_t col = it->first.first;
    int16_t line = it->first.second;

    strong_pixel_maps[i].erase(it); // Remove strong pixel from the dictionary, so it is not processed again

    ret_value.x = col * (double)photons; // position is weighted by number of photon counts
    ret_value.y = (line + (i%2) * LINES) * (double)photons;
    // Y accounts for the actual module
    ret_value.z = (i / 2) * (double)photons;
    ret_value.photons = photons;
    ret_value.pixels = 1;
    ret_value.depth = 0;

    strong_pixel_map_t::iterator it2;

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col-1, line  ))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col-1, line+1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col-1, line-1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col+1, line  ))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col+1, line-1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col+1, line+1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col  , line-1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if ((it2 = strong_pixel_maps[i].find(coordxy_t(col  , line+1))) != strong_pixel_maps[i].end())
        merge_spots(ret_value, add_pixel(strong_pixel_maps, i, it2, connect_frames));

    if (connect_frames && (i + 2 < strong_pixel_maps.size())) {
        if ((it2 = strong_pixel_maps[i+2].find(coordxy_t(col  , line))) != strong_pixel_maps[i+2].end())
            merge_spots_new_frame(ret_value, add_pixel(strong_pixel_maps, i+2, it2, connect_frames));
    }
    return ret_value;
}

void analyze_spots(strong_pixel *host_out, std::vector<spot_t> &spots, bool connect_frames, size_t images, size_t image0) {
    // key is location of strong pixel - value is number of photons
    // there is one mpa per fragment analyzed by GPU (2 horizontally connected modules)
    strong_pixel_maps_t strong_pixel_maps = strong_pixel_maps_t(images*2); 

    // Transfer strong pixels into dictionary
    for (size_t i = 0; i < images*2; i++) {
        size_t addr = i * MAX_STRONG;
        int k = 0;
        // There is maximum MAX_STRONG pixels
        // GPU kernel sets col to -1 for next element after last strong pixel
        // Photons equal zero could mean that kernel was not at all executed
        while ((k < MAX_STRONG) && (host_out[addr + k].col >= 0) && (host_out[addr + k].line >= 0) && (host_out[addr+k].photons > 0)) {
              coordxy_t key = coordxy_t(host_out[addr + k].col, host_out[addr + k].line);
              strong_pixel_maps[i][key] = host_out[addr + k].photons;
              k++;
        }
    }

    for (int i = 0; i < images*2; i++) {
      strong_pixel_map_t::iterator iterator = strong_pixel_maps[i].begin();
      while (iterator != strong_pixel_maps[i].end()) {
          spot_t spot = add_pixel(strong_pixel_maps, i, iterator, connect_frames);
          // Apply pixel count cut-off and cut-off of number of frames, which spot can span 
          // (spots present in most frames, are likely to be either bad pixels or in spindle axis)
          spot.x = spot.x / spot.photons;
          // Account for the fact, that each process handles only part of the detector
          spot.y = spot.y / spot.photons + (NCARDS - receiver_settings.card_number - 1) * 2 * LINES;
          // Account for frame number
          spot.z = spot.z / spot.photons + image0;
//          spots.push_back(spot);
          if ((spot.pixels > 3) && (spot.depth < 100)) spots.push_back(spot);
          iterator = strong_pixel_maps[i].begin(); // Get first unprocessed spot in this frame
      }
    }
}

void *run_gpu_thread(void *in_threadarg) {
    ThreadArg *arg = (ThreadArg *) in_threadarg;

    // GPU device is valid on per-thread basis, so every thread needs to set it
    hipSetDevice(receiver_settings.gpu_device);

    std::vector<spot_t> spots;

    // Account for leftover
    size_t total_chunks = experiment_settings.nimages_to_write / NIMAGES_PER_STREAM;
    if (experiment_settings.nimages_to_write - total_chunks * NIMAGES_PER_STREAM > 0)
           total_chunks++;

    size_t gpu_slice = arg->ThreadID;

    hipEvent_t event_mem_copied;
    hipEventCreate (&event_mem_copied);

    strong_pixel *host_out = (strong_pixel *) calloc(NIMAGES_PER_STREAM * 2 * MAX_STRONG, sizeof(strong_pixel));

    for (size_t chunk = gpu_slice;
         chunk < total_chunks;
         chunk += NCUDA_STREAMS) {

         size_t ib_slice = chunk % (NCUDA_STREAMS*CUDA_TO_IB_BUFFER);

//         size_t frame0 = ib_slice * NIMAGES_PER_STREAM;
         size_t images = experiment_settings.nimages_to_write - gpu_slice * NIMAGES_PER_STREAM;
         if (images > NIMAGES_PER_STREAM) images = NIMAGES_PER_STREAM;

         pthread_mutex_lock(writer_threads_done_mutex+ib_slice);
         // Wait till everyone is done
         while (writer_threads_done[ib_slice] < receiver_settings.compression_threads)
             pthread_cond_wait(writer_threads_done_cond+ib_slice, 
                               writer_threads_done_mutex+ib_slice);
         // Restore full values and continue
         writer_threads_done[ib_slice] = 0;
         pthread_mutex_unlock(writer_threads_done_mutex+ib_slice);

         // Here all writting is done, but it is guarranteed not be overwritten

         // Copy frames to GPU memory
         hipError_t err;
         err = hipMemcpyAsync(gpu_data16 + (gpu_slice % NCUDA_STREAMS) * NIMAGES_PER_STREAM * FRAME_SIZE / sizeof(uint16_t), 
               ib_buffer + ib_slice * NIMAGES_PER_STREAM * FRAME_SIZE,
               images * FRAME_SIZE,
               hipMemcpyHostToDevice, stream[gpu_slice]);
         if (err != hipSuccess) {
             std::cerr << "GPU: memory copy error for slice " << gpu_slice << "/" << ib_slice << "frames: " << images << "(" << hipGetErrorString(err) << ")" << std::endl;
             pthread_exit(0);
         }

         hipEventRecord (event_mem_copied, stream[gpu_slice]);

         // Start GPU kernel
         // TODO - handle frame summation
         find_spots_colspot<int16_t> <<<NIMAGES_PER_STREAM * 2 / 32, 32, 0, stream[gpu_slice]>>> 
                 (gpu_data16 + gpu_slice * NIMAGES_PER_STREAM * FRAME_SIZE / 2, 
                  gpu_out + gpu_slice * NIMAGES_PER_STREAM * 2 * MAX_STRONG, 
                  experiment_settings.strong_pixel, images * 2);

         // After data are copied, one can release buffer
         err = hipEventSynchronize(event_mem_copied);
         if (err != hipSuccess) {
             std::cerr << "GPU: memory copy error" << std::endl;
             pthread_exit(0);
         }

         // Broadcast to everyone waiting, that buffer can be overwritten by next iteration
         pthread_mutex_lock(cuda_stream_ready_mutex+ib_slice);
         cuda_stream_ready[ib_slice] = chunk + NCUDA_STREAMS*CUDA_TO_IB_BUFFER;
         pthread_cond_broadcast(cuda_stream_ready_cond+ib_slice);
         pthread_mutex_unlock(cuda_stream_ready_mutex+ib_slice);

         // Ensure kernel has finished
         err = hipStreamSynchronize(stream[gpu_slice]);
         if (err != hipSuccess) {
             std::cerr << "GPU: execution error" << std::endl;
             pthread_exit(0);
         }

         // Copy result back to host memory
         err = hipMemcpy(host_out, 
                         gpu_out + gpu_slice * NIMAGES_PER_STREAM * 2 * MAX_STRONG,
                         images * 2 * MAX_STRONG * sizeof(strong_pixel),
                         hipMemcpyDeviceToHost);

         // Analyze results to find spots
         analyze_spots(host_out, spots, experiment_settings.connect_spots_between_frames, images, chunk * NIMAGES_PER_STREAM);
    }
    hipEventDestroy (event_mem_copied);

    // Merge calculated spots to a single vector
    pthread_mutex_lock(&all_spots_mutex);
    for (int i = 0; i < spots.size(); i++)
        all_spots.push_back(spots[i]);
    pthread_mutex_unlock(&all_spots_mutex);

//    std::cout << "GPU: Thread "<< arg->ThreadID << " done" << std::endl;
    pthread_exit(0);
}

